#include "hip/hip_runtime.h"
#include "ATen/cuda/HIPContext.h"
#include <ATen/ATen.h>
#include <torch/torch.h>
#include <torch/types.h>
namespace at {
namespace native {

namespace {
template <typename scalar_t>
__global__ void revert_varlen_kernel(scalar_t *in, scalar_t *out,
                                     int64_t *offsets, int feature_size, int n,
                                     scalar_t pad_value) {
  const int offset = static_cast<int>(offsets[blockIdx.x]);
  for (int i = threadIdx.x; i < feature_size; i += blockDim.x) {
    out[blockIdx.x * feature_size + i] =
        (offset >= 0) ? in[offset + i] : pad_value;
  }
}

} // namespace

void checkLongTensor(const Tensor &tensor) {
  TORCH_CHECK(tensor.dim() == 1 && tensor.device() == at::kCPU &&
                  tensor.scalar_type() == at::kLong,
              "'lengths' argument should be a 1D CPU int64 tensor");
}

at::Tensor revert_varlen_tensor(const Tensor &_input, const Tensor &_offsets) {
  auto input = _input.contiguous();
  auto output = torch::empty_like(input);
  int64_t seq_length = input.size(0);
  int64_t batch_size = input.size(1);

  assert(_offsets.dim() == 1);
  assert(_offsets.is_cuda());
  assert(_offsets.scalar_type() == at::kLong);
  TORCH_CHECK(_offsets.dim() == 1 && _offsets.is_cuda() &&
                  _offsets.scalar_type() == at::kLong,
              "'offsets' argument should be a 1D CUDA int64 tensor");
  TORCH_CHECK(_offsets.numel() == batch_size * seq_length,
              "Expected `len(offsets) = batch_size * seq_length`, but got ",
              _offsets.numel(), " (batch_size=", batch_size,
              ", seq_length=", seq_length, ")");

  int64_t feature_size = 1;
  for (int64_t dim = 2; dim < input.ndimension(); dim++) {
    feature_size *= input.size(dim);
  }

  int numThreads = 512;
  int numBlocks = batch_size * seq_length;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "revert_varlen", [&] {
        revert_varlen_kernel<<<numBlocks, numThreads, 0,
                               at::cuda::getCurrentCUDAStream()>>>(
            input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
            _offsets.data_ptr<int64_t>(), feature_size, batch_size * seq_length,
            static_cast<scalar_t>(0));
      });

  return output;
}

at::Tensor get_offsets(const Tensor &_input, const Tensor &_lengths) {
  at::native::checkLongTensor(_lengths);
  auto input = _input.contiguous();
  int64_t seq_length = input.size(0);
  int64_t batch_size = input.size(1);
  int64_t *lengths = _lengths.data_ptr<int64_t>();

  TORCH_CHECK(_lengths.size(0) == batch_size,
              "Expected `len(lengths)` to be equal to batch_size, but got ",
              _lengths.size(0), " (batch_size=", batch_size, ")");
  TORCH_CHECK(
      (lengths[batch_size - 1] > 0),
      "Length of all samples has to be greater than 0, but found an element "
      "in 'lengths' that is <= 0");

  std::vector<int64_t> offsets;
  offsets.reserve(batch_size * seq_length);
  int64_t feature_size = 1;
  for (int64_t dim = 2; dim < input.ndimension(); dim++) {
    feature_size *= input.size(dim);
  }
  for (int64_t t = 0; t < seq_length; t++) {
    for (int64_t i = 0; i < batch_size; i++) {
      if (lengths[i] > t) {
        offsets.push_back(i * feature_size +
                          (lengths[i] - t - 1) * batch_size * feature_size);
      } else {
        offsets.push_back(-1);
      }
    }
  }

  auto options = at::TensorOptions().device(at::kCUDA).dtype(at::kLong);
  auto offsets_tensor =
      at::from_blob(offsets.data(), batch_size * seq_length, at::kLong)
          .to(options, /* non_blocking */ true, /*copy*/ false);
  return offsets_tensor;
}

} // namespace native
} // namespace at
